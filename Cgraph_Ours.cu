#include "hip/hip_runtime.h"
#include <iostream>

// #define n 13
// #define m 23
using namespace std;
#include <vector>
#include <algorithm>


__global__ void dummyKernel() {
    // Placeholder kernel
}

int main() {
    int n=13;
    int m=23;

    // Example initialization (optional)
    int ver[n]={0,5,8,8,9,9,10,14,15,18,19,21,23};
    int edges[m]={1,5,8,3,7,9,5,10,6,0,0,4,8,5,5,2,4,11,5,2,8,5,2};
    int weight[m]={2,4,9,1,8,1,2,5,2,4,3,5,2,4,3,1,2,4,1,6,1};

    bfs(ver, edges, 1); // Call BFS with the first vertex as source

    std::vector<int> sink(n, 0); // Initialize all elements to 0

    sink.clear(); // Clear the vector to ensure it's empty before pushing back
    for (int i = 0; i < n - 1; i++) {
        if (ver[i] == ver[i + 1]) {
            sink.push_back(i);
        }
    }


    int count[m] = {0}; // Initialize all elements to 0
    for (int i = 0; i < m; i++) {
        count[edges[i]]++;
    }

    for (size_t i = 0; i < sink.size(); i++) {
        int sinkIndex = sink[i];
        if (sinkIndex >= 0 && sinkIndex < n) {
            for (int j = sinkIndex; j < m - 1; j++) {
                count[j] = count[j + 1]; // Shift elements to the left
            }
            count[m - 1] = 0; // Set the last element to 0 after shifting
            m--; // Reduce the size of the array
        }
    }

    std::vector<std::pair<int, int>> countPairs;

    for (int i = 0; i < n; i++) {
        countPairs.push_back(std::make_pair(count[i], i));
    }

    // Sort the pairs based on the first value in reverse order
    std::sort(countPairs.begin(), countPairs.end(), [](const std::pair<int, int>& a, const std::pair<int, int>& b) {
        return a.first > b.first;
    });

    // Print the sorted pairs
    for (const auto& p : countPairs) {
        std::cout << "count: " << p.first << ", index: " << p.second << std::endl;
    }

    // Print the array count
    for (int i = 0; i < n; i++) {
        std::cout << "count[" << i << "] = " << count[i] << std::endl;
    }

    for (size_t i = 0; i < sink.size(); i++) {
        std::cout << "sink[" << i << "] = " << sink[i] << std::endl;
    }

    // Placeholder for further implementation
    std::cout << "Arrays initialized." << std::endl;

    return 0;
}